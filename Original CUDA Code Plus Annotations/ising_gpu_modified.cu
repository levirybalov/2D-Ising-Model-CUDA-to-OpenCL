#include "hip/hip_runtime.h"
/****
 *
 * GPU accelerated Monte Carlo simulation of the 2D Ising model
 *
 * Copyright (C) 2008 Tobias Preis (http://www.tobiaspreis.de)
 *
 * This program is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * as published by the Free Software Foundation; either version
 * 3 of the License, or (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public
 * License along with this program; if not, see
 * http://www.gnu.org/licenses/.
 *
 * Related publication:
 *
 * T. Preis, P. Virnau, W. Paul, and J. J. Schneider,
 * Journal of Computational Physics 228, 4468-4477 (2009)
 * doi:10.1016/j.jcp.2009.03.018
 *
 */

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <cutil.h>

#define FLAG_PRINT_SPINS 0
// define FLAG_ENERGY 1 to calculate energy, FLAG_ENERGY 0 to calculate magnetization
#define FLAG_ENERGY 1
#define T_START 3.00
#define T_FACTOR 0.9
#define T_END 2.00
#define GLOBAL_ITERATIONS 100
#define RANDOM_A 1664525
#define RANDOM_B 1013904223

#define BLOCK_SIZE 256

// n = one side of the lattice; N = number of lattice sites
const unsigned int N=4*BLOCK_SIZE*BLOCK_SIZE;
const unsigned int n=2*BLOCK_SIZE;

/****
 *
 *  Function declaration
 *
 */
void calc(int argc,char** argv);
void cpu_function(double*,int*);
__global__ void device_function_main(int*,int*,int*,float,bool);

/****
 *
 *  Main function
 *
 */
int main(int argc,char** argv) {
  calc(argc,argv);
}

/****
 *
 *  Calc
 *
 */
void calc(int argc,char** argv) {

  printf(" ----------------------------------------------------------------------- \n");
  printf(" *\n");
  printf(" *  GPU accelerated Monte Carlo simulation of the 2D Ising model\n");
  printf(" *\n");
  printf(" *  Copyright (C) 2008 Tobias Preis (http://www.tobiaspreis.de)\n");
  printf(" *\n");
  printf(" *  This program is free software; you can redistribute it and/or\n");
  printf(" *  modify it under the terms of the GNU General Public License\n");
  printf(" *  as published by the Free Software Foundation; either version\n");
  printf(" *  3 of the License, or (at your option) any later version.\n");
  printf(" *\n");
  printf(" *  This program is distributed in the hope that it will be useful,\n");
  printf(" *  but WITHOUT ANY WARRANTY; without even the implied warranty of\n");
  printf(" *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the\n");
  printf(" *  GNU General Public License for more details.\n");
  printf(" *\n");
  printf(" *  You should have received a copy of the GNU General Public\n");
  printf(" *  License along with this program; if not, see\n");
  printf(" *  http://www.gnu.org/licenses/\n");
  printf(" *\n");
  printf(" *  Related publication:\n");
  printf(" *\n");
  printf(" *  T. Preis, P. Virnau, W. Paul, and J. J. Schneider,\n");
  printf(" *  Journal of Computational Physics 228, 4468-4477 (2009)\n");
  printf(" *  doi:10.1016/j.jcp.2009.03.018\n");
  printf(" *\n");

  printf(" ----------------------------- Ising model ----------------------------- \n");
  printf(" Number of Spins: %d \n",N);
  printf(" Start Temperature: %f \n",T_START);
  printf(" Decreasing Factor: %f \n",T_FACTOR);
  printf(" Final Temperature: %f \n",T_END);
  printf(" Global Iterations: %d \n",GLOBAL_ITERATIONS);

  //Init
  CUT_DEVICE_INIT(argc,argv);
  srand48(23);

  //Allocate and init host memory for output arrays
  // h_ for host
  int num_entries=0;
  // the following loop makes num_entries = the number of temperatures at which
  // the monte carlo simulation is run
  for(double t=T_START; t>=T_END; t=t*T_FACTOR) num_entries++;
  // mem_out_size is the size of the output arrays for temperatures and energies,
  // which obviously should be a float
  unsigned int mem_out_size=sizeof(float)*num_entries;
  // h_T is the host (final) output array for the Temperatures
  float* h_T=(float*) malloc(mem_out_size);
  // h_E is the host (final) output array for the Energies
  float* h_E=(float*) malloc(mem_out_size);
  // mem_ref_out_size is an array of size num_entries, each entry is a double
  unsigned int mem_ref_out_size=sizeof(double)*num_entries;
  // h_ref_E is the reference array for the CPU
  double* h_ref_E=(double*) malloc(mem_ref_out_size);
  // the following loop fills h_T with the appropriate temperatures
  num_entries=0;
  for(double t=T_START; t>=T_END; t=t*T_FACTOR) {
    h_T[num_entries]=t;
    num_entries++;
  }

  //Allocate and init host memory for simulation arrays
  unsigned int mem_size=sizeof(int)*N;
  unsigned int mem_size_random=sizeof(int)*BLOCK_SIZE*BLOCK_SIZE;
  // h_random_data is an integer array of size mem_size_random, meant
  // to hold BLOCK_SIZE random numbers for each of the BLOCK_SIZE blocks
  int* h_random_data=(int*) malloc(mem_size_random);
  // h_S is an integer array of size mem_size meant to hold the Spins
  int* h_S=(int*) malloc(mem_size);
  unsigned int mem_size_out=sizeof(int)*BLOCK_SIZE;
  // h_out is an integer array  of size mem_size_out that holds the REDUCED energies,
  // which is why it has mem_size_out elements, rather than N elements
  int* h_out=(int*) malloc(mem_size_out);
  // the following loop fills h_random_data with consecutive powers of 16807
  // according to eqn. 2 in the paper; the mod operation is done in the GPU kernel
  // during the spin flip determination
  h_random_data[0]=1;
  for(int i=1;i<BLOCK_SIZE*BLOCK_SIZE;i++) {
    h_random_data[i]=16807*h_random_data[i-1];
  }
  // the following loop initializes h_S to spin values of +/- 1 with equal
  // probability
  for(int i=0;i<N;i++) {
    if(drand48()>0.5) h_S[i]=-1;
    else h_S[i]=1;
  }

  //Create and start timer
  // CUDA_SAFE_CALL is outdated and unnecessary as of a while ago (CUDA 5.0?)
  // and it can pretty much just be regarded as calling the function that is its
  // argument
  float gpu_sum=0;
  unsigned int timer=0;
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  CUT_SAFE_CALL(cutCreateTimer(&timer));
  CUT_SAFE_CALL(cutStartTimer(timer));

  //Allocate device memory for arrays
  // d_ for device
  int* d_random_data;
  int* d_S;
  int* d_out;
  // creates unintialized arrays on the GPU corresponding to the arrays on the CPU from
  // which they are being copied
  CUDA_SAFE_CALL(hipMalloc((void**) &d_random_data,mem_size_random));
  CUDA_SAFE_CALL(hipMalloc((void**) &d_S,mem_size));
  CUDA_SAFE_CALL(hipMalloc((void**) &d_out,mem_size_out));

  //Stop and destroy timer
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  CUT_SAFE_CALL(cutStopTimer(timer));
  float gpu_dt_malloc=cutGetTimerValue(timer);
  gpu_sum+=gpu_dt_malloc;
  printf("\n --------------------------------- GPU --------------------------------- \n");
  printf(" Processing time on GPU for allocating: %f (ms) \n",gpu_dt_malloc);
  CUT_SAFE_CALL(cutDeleteTimer(timer));

  //Create and start timer
  timer=0;
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  CUT_SAFE_CALL(cutCreateTimer(&timer));
  CUT_SAFE_CALL(cutStartTimer(timer));

  //Copy host memory to device and create mirror of d_S
  CUDA_SAFE_CALL(hipMemcpy(d_random_data,h_random_data,mem_size_random,hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(d_S,h_S,mem_size,hipMemcpyHostToDevice));

  //Stop and destroy timer
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  CUT_SAFE_CALL(cutStopTimer(timer));
  float gpu_dt_mem=cutGetTimerValue(timer);
  gpu_sum+=gpu_dt_mem;
  printf(" Processing time on GPU for memory transfer: %f (ms) \n",gpu_dt_mem);
  CUT_SAFE_CALL(cutDeleteTimer(timer));

  //Print spins
  if(FLAG_PRINT_SPINS) {
    // copy spins from device to host
    CUDA_SAFE_CALL(hipMemcpy(h_S,d_S,mem_size,hipMemcpyDeviceToHost));
    for(int i=0;i<BLOCK_SIZE*2;i++) {
      for(int j=0;j<BLOCK_SIZE*2;j++) {
  // this prints a row with BLOCK_SIZE*2 columns
	if(h_S[i*BLOCK_SIZE*2+j]>0) printf("+ ");
	else printf("- ");
      }
      printf("\n");
    }
    printf("\n");
  }

  //Create and start timer
  timer=0;
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  CUT_SAFE_CALL(cutCreateTimer(&timer));
  CUT_SAFE_CALL(cutStartTimer(timer));

  //Calc energy
  num_entries=0;
  dim3 threads(BLOCK_SIZE);
  dim3 grid(BLOCK_SIZE);
  for(float t=T_START;t>=T_END;t=t*T_FACTOR) {
    double avg_H=0;
    for(int global_iteration=0;global_iteration<GLOBAL_ITERATIONS;global_iteration++) {
      device_function_main<<<grid,threads>>>(d_S,d_out,d_random_data,t,true);
      device_function_main<<<grid,threads>>>(d_S,d_out,d_random_data,t,false);

      CUDA_SAFE_CALL(hipMemcpy(h_out,d_out,mem_size_out,hipMemcpyDeviceToHost));
      int energy_sum=0;
      for(int i=0;i<BLOCK_SIZE;i++) energy_sum+=h_out[i];
      avg_H+=(float)energy_sum/N;
    }
    h_E[num_entries]=avg_H/GLOBAL_ITERATIONS;
    num_entries++;
  }

  //Stop and destroy timer
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  CUT_SAFE_CALL(cutStopTimer(timer));
  float gpu_dt_main=cutGetTimerValue(timer);
  gpu_sum+=gpu_dt_main;
  printf(" Processing time on GPU for main function: %f (ms) \n",gpu_dt_main);
  printf(" Total processing time on GPU: %f (ms) \n",gpu_sum);
  CUT_SAFE_CALL(cutDeleteTimer(timer));

  //Check kernel execution
  CUT_CHECK_ERROR("Kernel execution failed");

  //Print spins
  if(FLAG_PRINT_SPINS) {
    CUDA_SAFE_CALL(hipMemcpy(h_S,d_S,mem_size,hipMemcpyDeviceToHost));
    for(int i=0;i<BLOCK_SIZE*2;i++) {
      for(int j=0;j<BLOCK_SIZE*2;j++) {
	if(h_S[i*BLOCK_SIZE*2+j]>0) printf("+ ");
	else printf("- ");
      }
      printf("\n");
    }
  }

  //Create and start timer
  timer=0;
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  CUT_SAFE_CALL(cutCreateTimer(&timer));
  CUT_SAFE_CALL(cutStartTimer(timer));

  //Reference solution
  cpu_function(h_ref_E,h_S);

  //Print spins
  if(FLAG_PRINT_SPINS) {
    printf("\n");
    for(int i=0;i<BLOCK_SIZE*2;i++) {
      for(int j=0;j<BLOCK_SIZE*2;j++) {
	if(h_S[i*BLOCK_SIZE*2+j]>0) printf("+ ");
	else printf("- ");
      }
      printf("\n");
    }
  }

  //Stop and destroy timer
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  CUT_SAFE_CALL(cutStopTimer(timer));
  float cpu_sum=cutGetTimerValue(timer);
  printf("\n --------------------------------- CPU --------------------------------- \n");
  printf(" Total processing time on CPU: %f (ms) \n",cpu_sum);
  CUT_SAFE_CALL(cutDeleteTimer(timer));
  printf("\n Speedup: %fX \n\n",(cpu_sum/gpu_sum));

  //Cleaning memory
  free(h_T);
  free(h_E);
  free(h_ref_E);
  free(h_random_data);
  free(h_S);
  free(h_out);
  CUDA_SAFE_CALL(hipFree(d_random_data));
  CUDA_SAFE_CALL(hipFree(d_S));
  CUDA_SAFE_CALL(hipFree(d_out));
}

/****
 *
 *  Device function main
 *
 */
__global__ void device_function_main(int* S,int* out,int* R,float t,bool flag) {

  //Energy variable
  // there is a dH for every thread in every block
  int dH=0;
  float exp_dH_4=exp(-(4.0)/t);
  float exp_dH_8=exp(-(8.0)/t);

  //Allocate shared memory
  // shared memmory is allocated per thread block
  __shared__ int r[BLOCK_SIZE];

  //Load random data
  // recall, as just mentioned above, r is allocated per thread block
  r[threadIdx.x]=R[threadIdx.x+BLOCK_SIZE*blockIdx.x];
  __syncthreads();

  // first part of first conditional -> will hit if flag == true; executes steps (a) and (b) described in the paper
  if(flag) {

    // the top left site of each 2x2 cell are dealt with in the following order:
    // 1) top left cell
    // 2) top cells that aren't the leftmost
    // 3) leftmost cells that aren't the top
    // 4) all other top left cells

    //Create new random numbers
    r[threadIdx.x]=RANDOM_A*r[threadIdx.x]+RANDOM_B;

    //Spin update top left
    if(blockIdx.x==0) { //Top
      if(threadIdx.x==0) { //Left
        // so here we are just accessing the top left site
        dH=2*S[2*threadIdx.x]*(
                               S[2*threadIdx.x+1]+ // site to the right
                               S[2*threadIdx.x-1+2*BLOCK_SIZE]+ // site to the "left" (wrapping around)
                               S[2*threadIdx.x+2*BLOCK_SIZE]+ // site below; notice that these are linear indices
                                                              // so this site and the previous one only differ by 1
                               S[2*threadIdx.x+N-2*BLOCK_SIZE]); // site "above" (wrapping around)
      }
      // top row minus leftmost cell
      else {
        dH=2*S[2*threadIdx.x]*(
                               S[2*threadIdx.x+1]+ // site to the left
                               S[2*threadIdx.x-1]+ // site to the right
                               S[2*threadIdx.x+2*BLOCK_SIZE]+ // site below
                               S[2*threadIdx.x+N-2*BLOCK_SIZE]); // site "above" (wrapping around)
      }
    }
    else { // not top
      if(threadIdx.x==0) { //Left
        // index below implies that we are accessing sites 4*BLOCK_SIZE, 8*BLOCK_SIZE, 12*BLOCK_SIZE, ...
        // so leftmost column, but not top point
        dH=2*S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x]*(
                                                       S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+1]+ // site to the right
                                                       S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x-1+2*BLOCK_SIZE]+ // site to the "left" (wrapping around)
                                                       S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]+ // site below
                                                       S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x-2*BLOCK_SIZE]); // site above
      }
      else {
        // all other cells
        dH=2*S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x]*(
                                                       S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+1]+ // site to the right
                                                       S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x-1]+ // site to the left
                                                       S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]+ // site below
                                                       S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x-2*BLOCK_SIZE]); // site above
      }
    }

    // if Hamiltonian == 4
    if(dH==4) {
      if(fabs(r[threadIdx.x]*4.656612e-10)<exp_dH_4) {
        S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x]=-S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x];
      }
    }
    // if Hamiltonian == 8
    else if(dH==8) {
      if(fabs(r[threadIdx.x]*4.656612e-10)<exp_dH_8) {
        S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x]=-S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x];
      }
    }
    // if Hamiltonian <= 0
    else {
      S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x]=-S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x];
    }

    //Create new random numbers
    r[threadIdx.x]=RANDOM_A*r[threadIdx.x]+RANDOM_B;

    //Spin update bottom right
    // notice that indices are offest by a constant of 2*BLOCK_SIZE;
    // first linear index here will be 513, so bottom right of top left square, as expected
    if(blockIdx.x==BLOCK_SIZE-1) { //Bottom
      if(threadIdx.x==BLOCK_SIZE-1) { //Right
        dH=2*S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]*(
                                                                      S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2]+ // site to the "right" (wrapping around)
                                                                      S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]+  // site to the left
                                                                      S[2*threadIdx.x+1]+ // site "below" (wrapping around)
                                                                      S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x]); // site above (2*BLOCK_SIZE removed)
      }
      else {
        dH=2*S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]*(
                                                                      S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE+2]+ // site to the right
                                                                      S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]+ // site to the left
                                                                      S[2*threadIdx.x+1]+ // site "below" (wrapping around)
                                                                      S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x]); // site above (2*BLOCK_SIZE removed)
      }
    }
    else {
      if(threadIdx.x==BLOCK_SIZE-1) { //Right
        dH=2*S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]*(
                                                                      S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2]+ // site to the "right" (wrapping around)
                                                                      S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]+ // site to the left
                                                                      S[2*threadIdx.x+1+4*BLOCK_SIZE*(blockIdx.x+1)]+ // site below (2*BLOCK_SIZE added)
                                                                      S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x]); // site above (2*BLOCK_SIZE removed)
      }
      else {
        dH=2*S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]*(
                                                                      S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE+2]+ // site to the right
                                                                      S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]+ // site to the left
                                                                      S[2*threadIdx.x+1+4*BLOCK_SIZE*(blockIdx.x+1)]+ // site below (2*BLOCK_SIZE added)
                                                                      S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x]); // site above (2*BLOCK_SIZE removed)
      }
    }

    if(dH==4) {
      if(fabs(r[threadIdx.x]*4.656612e-10)<exp_dH_4) {
        S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]=-S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE];
      }
    }
    else if(dH==8) {
      if(fabs(r[threadIdx.x]*4.656612e-10)<exp_dH_8) {
        S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]=-S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE];
      }
    }
    else {
      S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]=-S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE];
    }

    __syncthreads();

  }
  // second part of first conditional -> will hit if flag == false; will execute steps (c) and (d)
  else {

    //Create new random numbers
    r[threadIdx.x]=RANDOM_A*r[threadIdx.x]+RANDOM_B;

    //Spin update top right
    if(blockIdx.x==0) { //Top
      if(threadIdx.x==BLOCK_SIZE-1) { //Right
        dH=2*S[2*threadIdx.x+1]*(
                                 S[2*threadIdx.x+2-2*BLOCK_SIZE]+ // site to the "right". wrapping around
                                 S[2*threadIdx.x]+ // site to the left
                                 S[2*threadIdx.x+1+2*BLOCK_SIZE]+ // site below
                                 S[2*threadIdx.x+1+N-2*BLOCK_SIZE]); // site "above", wrapping around
      }
      else {
        dH=2*S[2*threadIdx.x+1]*(
                                 S[2*threadIdx.x+2]+ // site to the right
                                 S[2*threadIdx.x]+ // site to the left
                                 S[2*threadIdx.x+1+2*BLOCK_SIZE]+ // site below
                                 S[2*threadIdx.x+1+N-2*BLOCK_SIZE]); // site "above", wrapping around
      }
    }
    else {
      if(threadIdx.x==BLOCK_SIZE-1) { //Right
        dH=2*S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x]*(
                                                         S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2-2*BLOCK_SIZE]+ // site to the "right", wrapping around
                                                         S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x]+ // site to the left
                                                         S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]+ // site below
                                                         S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x-2*BLOCK_SIZE]); // site above
      }
      else {
        dH=2*S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x]*(
                                                         S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2]+ // site to the right
                                                         S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x]+ // site to the left
                                                         S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]+ // site below
                                                         S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x-2*BLOCK_SIZE]); //site above
      }
    }

    if(dH==4) {
      if(fabs(r[threadIdx.x]*4.656612e-10)<exp_dH_4) {
        S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x]=-S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x];
      }
    }
    else if(dH==8) {
      if(fabs(r[threadIdx.x]*4.656612e-10)<exp_dH_8) {
        S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x]=-S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x];
      }
    }
    else {
      S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x]=-S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x];
    }

    //Create new random numbers
    r[threadIdx.x]=RANDOM_A*r[threadIdx.x]+RANDOM_B;

    //Spin update bottom left
    if(blockIdx.x==BLOCK_SIZE-1) { //Bottom
      if(threadIdx.x==0) { //Left
        dH=2*S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]*(
                                                                    S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE+1]+ // site to the right
                                                                    S[2*threadIdx.x+4*BLOCK_SIZE*(blockIdx.x+1)-1]+ // site to the "left" (wrapping around)
                                                                    S[2*threadIdx.x]+ // site "below" (wrapping around)
                                                                    S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x]); // site above
      }
      else {
        dH=2*S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]*(
                                                                    S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE+1]+ // site to the right
                                                                    S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE-1]+ // site to the left
                                                                    S[2*threadIdx.x]+ // site "below" (wrapping around)
                                                                    S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x]); // site above
      }
    }
    else {
      if(threadIdx.x==0) { //Left
        dH=2*S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]*(
                                                                    S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE+1]+ // site to the right
                                                                    S[2*threadIdx.x+4*BLOCK_SIZE*(blockIdx.x+1)-1]+ // site to the "left" (wrapping around)
                                                                    S[2*threadIdx.x+4*BLOCK_SIZE*(blockIdx.x+1)]+ // site below
                                                                    S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x]); // site above
      }
      else {
        dH=2*S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]*(
                                                                    S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE+1]+ // site to the right
                                                                    S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE-1]+ // site to the left
                                                                    S[2*threadIdx.x+4*BLOCK_SIZE*(blockIdx.x+1)]+ // site below
                                                                    S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x]); // site above
      }
    }

    if(dH==4) {
      if(fabs(r[threadIdx.x]*4.656612e-10)<exp_dH_4) {
        S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]=-S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE];
      }
    }
    else if(dH==8) {
      if(fabs(r[threadIdx.x]*4.656612e-10)<exp_dH_8) {
        S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]=-S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE];
      }
    }
    else {
      S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]=-S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE];
    }

  }

  //Transfer random data back to global memory
  // this is saving the current set of random numbers, so they are not reused
  // (they will be loaded and modified when this GPU kernel is called again)
  R[threadIdx.x+BLOCK_SIZE*blockIdx.x]=r[threadIdx.x];

  if(!flag) { // will hit if flag == false, i.e. after second set of spin updates

    //For reduction shared memory array r is used
    if(FLAG_ENERGY) {

      //Calc energy
      // recall that each 2x2 cell is represented by a single thread, which has its own dH
      if(blockIdx.x==BLOCK_SIZE-1) { //Bottom
	if(threadIdx.x==BLOCK_SIZE-1) { //Right
       // bottom right cell (last thread in last block):
       // top left site * (site to the right + site below)
	  dH=-S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x]*(S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+1]+S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE])
       // top right site * (site to the "right" (wrapping around) + site below)
	     -S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x]*(S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x+1-2*BLOCK_SIZE]+S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE])
       // bottom left site * (site to the right + site "below" (wrapping around))
	     -S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]*(S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+1+2*BLOCK_SIZE]+S[2*threadIdx.x])
       // bottom right site * (site to the "right" (wrapping around) + site "below" (wrapping around))
	     -S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]*(S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2]+S[2*threadIdx.x+1]);
	}
	else {
       // bottom block minus bottom right cell:
       // top left site * (site to the right + site below)
	  dH=-S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x]*(S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+1]+S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE])
       // top right site * (site to the right + site below)
       -S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x]*(S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x+1]+S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE])
       // bottom left site * (site to the right + site "below" (wrapping around))
	     -S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]*(S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+1+2*BLOCK_SIZE]+S[2*threadIdx.x])
       // bottom right site * (site to the right + site "below" (wrapping around))
	     -S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]*(S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2+2*BLOCK_SIZE]+S[2*threadIdx.x+1]);
	}
      }
      else {
	if(threadIdx.x==BLOCK_SIZE-1) { //Right
       // rightmost thread minus bottom right cell
       // top left site * (site to the right + site below)
	  dH=-S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x]*(S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+1]+S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE])
       // top right site * (site to the "right" (wrapping around) + site below)
       -S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x]*(S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x+1-2*BLOCK_SIZE]+S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE])
       // bottom left site * (site to the right + site below)
	     -S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]*(S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+1+2*BLOCK_SIZE]+S[2*threadIdx.x+4*BLOCK_SIZE*(blockIdx.x+1)])
       // bottom right site * (site to the "right" (wrapping around) + site below)
	     -S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]*(S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2]+S[2*threadIdx.x+1+4*BLOCK_SIZE*(blockIdx.x+1)]);
	}
	else {
       // all threads minus bottom block and rightmost threads
       // top left site * (site to the right + site below)
	  dH=-S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x]*(S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+1]+S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE])
       // top right site * (site to the right + site below)
	     -S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x]*(S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x+1]+S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE])
       // bottom left site * (site to the right + site below)
	     -S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]*(S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+1+2*BLOCK_SIZE]+S[2*threadIdx.x+4*BLOCK_SIZE*(blockIdx.x+1)])
       // bottom right site * (site to the right + site below)
	     -S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]*(S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2+2*BLOCK_SIZE]+S[2*threadIdx.x+1+4*BLOCK_SIZE*(blockIdx.x+1)]);
	}
      }
      __syncthreads();

    }
    else {

      //Calc magnetisation
      dH=S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x]
	+S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x]
	+S[2*threadIdx.x+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE]
	+S[2*threadIdx.x+1+4*BLOCK_SIZE*blockIdx.x+2*BLOCK_SIZE];
      __syncthreads();

    }

    //Save partial results back to shared memory in new structure
    // recall there is a dH for every thread, and an r for every block
    // and that dH is the reduced energy for each 2x2 cell
    r[threadIdx.x]=dH;

    //Reduction on GPU
    // "A binary tree structure realizes a fast reduction of the partial values within a block. These partial results of
    // each block are stored at block-dependent positions in global memory..."
    for(unsigned int dx=1;dx<BLOCK_SIZE;dx*=2) {
      if(threadIdx.x%(2*dx)==0) {
	r[threadIdx.x]+=r[threadIdx.x+dx];
      }
      __syncthreads();
    }

    //Save in out
    // partial results from above loop are stored in r[0] for each block
    if(threadIdx.x==0) out[blockIdx.x]=r[0];
  }
}

/****
 *
 *  CPU function
 *
 */
void cpu_function(double* E, int* S) {

  int random=23;
  int num_entries=0;

  for(double t=T_START;t>=T_END;t=t*T_FACTOR) {
    double avg_H=0;
    double exp_dH_4=exp(-(4.0)/t);
    double exp_dH_8=exp(-(8.0)/t);

    for(int global_iteration=0;global_iteration<GLOBAL_ITERATIONS;++global_iteration) {
      if(FLAG_ENERGY) {
	//Energy
	double H=0;
	for(int x=0;x<n;++x) {
	  for(int y=0;y<n;++y) {
	    int xr=x+1,yd=y+1;
	    if(xr==n) xr=0;
	    if(yd==n) yd=0;
	    H+=-S[y*n+x]*(S[y*n+xr]+S[yd*n+x]);
	  }
	}
	avg_H+=H/N;
      }
      else {
	//Magnetisation
	double H=0;
	for(int x=0;x<N;++x) {
	  H+=S[x];
	}
	avg_H+=H/N;
      }

      for(int x=0;x<n;++x) {
	for(int y=0;y<n;++y) {
    // first part of checkerboard
	  if((y*(n+1)+x)%2==0) {
	    int xl=x-1,yl=y,xu=x,yu=y-1,xr=x+1,yr=y,xd=x,yd=y+1;
	    if(x==0) {
	      xl=n-1;
	    }
	    else if(x==n-1) {
	      xr=0;
	    }
	    if(y==0) {
	      yu=n-1;
	    }
	    else if(y==n-1) {
	      yd=0;
	    }

	    //Initial local energy
	    int dH=2*S[y*n+x]*(
			       S[yl*n+xl]+
			       S[yr*n+xr]+
			       S[yu*n+xu]+
			       S[yd*n+xd]
			       );

	    if(dH==4) {
	      random=RANDOM_A*random+RANDOM_B;
	      if(fabs(random*4.656612e-10)<exp_dH_4) {
		S[y*n+x]=-S[y*n+x];
	      }
	    }
	    else if(dH==8) {
	      random=RANDOM_A*random+RANDOM_B;
	      if(fabs(random*4.656612e-10)<exp_dH_8) {
		S[y*n+x]=-S[y*n+x];
	      }
	    }
	    else {
	      S[y*n+x]=-S[y*n+x];
	    }
	  }
	}
      }

      for(int x=0;x<n;++x) {
	for(int y=0;y<n;++y) {
    // second part of checkerboard
	  if((y*(n+1)+x)%2==1) {
	    int xl=x-1,yl=y,xu=x,yu=y-1,xr=x+1,yr=y,xd=x,yd=y+1;
	    if(x==0) {
	      xl=n-1;
	    }
	    else if(x==n-1) {
	      xr=0;
	    }
	    if(y==0) {
	      yu=n-1;
	    }
	    else if(y==n-1) {
	      yd=0;
	    }

	    //Initial local energy
	    int dH=2*S[y*n+x]*(
			       S[yl*n+xl]+
			       S[yr*n+xr]+
			       S[yu*n+xu]+
			       S[yd*n+xd]
			       );

	    if(dH==4) {
	      random=RANDOM_A*random+RANDOM_B;
	      if(fabs(random*4.656612e-10)<exp_dH_4) {
		S[y*n+x]=-S[y*n+x];
	      }
	    }
	    else if(dH==8) {
	      random=RANDOM_A*random+RANDOM_B;
	      if(fabs(random*4.656612e-10)<exp_dH_8) {
		S[y*n+x]=-S[y*n+x];
	      }
	    }
	    else {
	      S[y*n+x]=-S[y*n+x];
	    }
	  }
	}
      }
    }
    E[num_entries]=avg_H/GLOBAL_ITERATIONS;
    num_entries++;
  }
}
